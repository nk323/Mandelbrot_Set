#include "hip/hip_runtime.h"
/* 
 * File:   MBSet.cu
 * 
 * Created on June 24, 2012
 * 
 * Purpose:  This program displays Mandelbrot set using the GPU via CUDA and
 * OpenGL immediate mode.
 * NAME: Neha Kadam
 * ECE 6122 Fall 2015 
 * 
 */

#include <iostream>
#include <stack>
#include <hip/hip_runtime_api.h>
#include <stdio.h>
#include "Complex.cu"

#include <GL/freeglut.h>

// Size of window in pixels, both width and height
#define WINDOW_DIM            512
#define NUM_THREADS 	      32
#define IMG_SIZE 	      WINDOW_DIM * WINDOW_DIM
#define NUM_BLOCKS 	      IMG_SIZE/ NUM_THREADS
#define DEBUG 		      0

using namespace std;

// Initial screen coordinates, both host and device.
Complex minC(-2.0, -1.2);
Complex maxC(1.0, 1.8);
Complex* dev_minC;
Complex* dev_maxC;
const int maxIt = 2000; // Maximum Iterations

Complex* c = new Complex[IMG_SIZE];//array to hold unique c values
int num_of_iterations[IMG_SIZE];

Complex* dev_c;  //c value array on the device
int* dev_iterations;

bool cudaMode = true;		
bool isSqrDrawn = false;
float dx, dy, diff;	//displacement variables
static int zoomLevel = 0;

// Function Declarations
void init();
void InitializeColors();
void drawMBSet();
void display();
void displayMBSet();

// Define the RGB Class
class RGB
{
public:
  RGB()
    : r(0), g(0), b(0) {}
  RGB(double r0, double g0, double b0)
    : r(r0), g(g0), b(b0) {}
public:
  double r;
  double g;
  double b;
};

RGB* colors = 0; // Array of color values

void InitializeColors()
{
  srand48(10);
  colors = new RGB[maxIt + 1];
  for (int i = 0; i < maxIt; ++i)
    {
      if (i < 6)
        { // Try this.. just white for small it counts
          colors[i] = RGB(1, 1, 1);
        }
      else
        {
	  colors[i] = RGB(drand48(), drand48(), drand48());
        }
    }
  colors[maxIt] = RGB(); // black
}

// Class to keep track of mouse click point
class Point
{
public:
  float x;
  float y;

  Point():x(0.0f), y(0.0f){}
};

Point start, end;

// Stack to store old values of minC and maxC when zooming in
stack< pair<Complex,Complex> > memStack;

// CUDA Function to compute MBSet 
__global__ void calcMB(Complex* dev_minC, Complex* dev_maxC, Complex* dev_c, int* dev_iterations)
{
  int id = threadIdx.x + blockIdx.x * blockDim.x;
  int x = id / WINDOW_DIM;
  int y = id % WINDOW_DIM;	

  double diffR = dev_maxC->r - dev_minC->r; //get diff between maxC and minC for real and imag
  double diffI = dev_maxC->i - dev_minC->i;
  double idR = (double) x / (WINDOW_DIM - 1); // to generate a unique c use the pixel location 
  double idI = (double) y / (WINDOW_DIM - 1);
  Complex newC = Complex(idR * diffR, idI * diffI);
  dev_c[id] = *dev_minC + newC;

  Complex Z(dev_c[id]);
  dev_iterations[id] = 0;
  
  // now compute Z till either iterations > maxIt or magnitude square > 4
  while(Z.magnitude2() < 4.0 && dev_iterations[id] < maxIt)
  {
    Z = (Z*Z) + dev_c[id];
    dev_iterations[id]++;
  }   

}
// Function to compute the Mandelbrot Set and act as wrapper for calling CUDA function
void drawMBSet()
{
  if(cudaMode)
  {
    if(DEBUG) cout << "Running in CUDA mode\n";
    // malloc
    hipMalloc((void**)&dev_minC, sizeof(Complex));
    hipMalloc((void**)&dev_maxC, sizeof(Complex));
    hipMalloc((void**)&dev_c, IMG_SIZE * sizeof(Complex));
    hipMalloc((void**)&dev_iterations, IMG_SIZE * sizeof(int));

    //now copy from host to device
    hipMemcpy(dev_minC, &minC, sizeof(Complex), hipMemcpyHostToDevice);
    hipMemcpy(dev_maxC, &maxC, sizeof(Complex), hipMemcpyHostToDevice);
    hipMemcpy(dev_c, c, IMG_SIZE * sizeof(Complex), hipMemcpyHostToDevice);
    hipMemcpy(dev_iterations, num_of_iterations, IMG_SIZE * sizeof(int), hipMemcpyHostToDevice);

    // call CUDA function
    calcMB<<< NUM_BLOCKS, NUM_THREADS >>>(dev_minC, dev_maxC, dev_c, dev_iterations);

    //now copy resuls from device to host
    hipMemcpy(c,dev_c, IMG_SIZE * sizeof(Complex), hipMemcpyDeviceToHost);
    hipMemcpy(num_of_iterations,dev_iterations, IMG_SIZE * sizeof(int), hipMemcpyDeviceToHost);
  }

  else // calculate without CUDA
  {
    int index = 0;

    for(int i = 0; i < WINDOW_DIM; i++)
    {
      for(int j = 0; j < WINDOW_DIM; j++)
      {
	index = i*WINDOW_DIM + j;
	
	double diffR = maxC.r - minC.r; //get diff between maxC and minC for real an imag
	double diffI = maxC.i - minC.i;
	double idR = (double) i / (WINDOW_DIM - 1); //generate unique c using pixel location 
	double idI = (double) j / (WINDOW_DIM - 1);
	Complex newC = Complex(idR * diffR, idI * diffI);
	c[index] = minC + newC;

	Complex Z(c[index]); // init Z0
	num_of_iterations[index] = 0;  // init number of iterations
		
	// now compute Z till either iterations > maxIt or magnitude sqaure > 4
	while(Z.magnitude2() < 4.0 && num_of_iterations[index] < maxIt)
	{
	  Z = (Z*Z) + c[index];
	  num_of_iterations[index]++;

	}
      }
    }

  }

}

void display()
{

  glClearColor(0.0, 0.0, 0.0, 1.0);
  glClear(GL_COLOR_BUFFER_BIT | GL_DEPTH_BUFFER_BIT);
  glLoadIdentity();
  gluLookAt(0.0, 0.0, 20.0, 0.0, 0.0, 0.0, 0.0, 1.0, 0.0); 
  displayMBSet();

  if(isSqrDrawn)
  {
    glColor3f(1.0, 0.0, 0.0);	//Red square
    
    glBegin(GL_LINE_LOOP);
      glVertex2f(start.x, start.y);	// x1,y1
      glVertex2f(end.x, start.y);	// x2,y1
      glVertex2f(end.x, end.y);		// x2,y2
      glVertex2f(start.x, end.y);	// x1,y2
    glEnd();
  }

  glutSwapBuffers();
  
}

void displayMBSet()
{

  int index = 0;
  glBegin(GL_POINTS);

  for(int i = 0; i < WINDOW_DIM; i++)
  {
    for(int j = 0; j < WINDOW_DIM; j++)
    {
	index = i*WINDOW_DIM + j;

	double r = colors[num_of_iterations[index]].r;	
	double g = colors[num_of_iterations[index]].g;		
	double b = colors[num_of_iterations[index]].b;	
	glColor3f(r,g,b);
	glVertex2i(i,j);

    }
  }

  glEnd();

}

void mouse(int button, int state, int x, int y)
{
  
  int i,j;
  
  if(button == GLUT_LEFT_BUTTON && state == GLUT_DOWN)
  {
      start.x = x;
      end.x = x;  
      
      start.y = y;
      end.y = y;
      
      isSqrDrawn = true;
  }

  if(button == GLUT_LEFT_BUTTON && state == GLUT_UP)
  {
      // push current minC and maxC values on stack
      memStack.push(make_pair(minC,maxC));

      //now calculate new minC and maxC based on zoom area
    	if(x > start.x)
    	{
      	  if(y > start.y)		// top left to bottom right
      	  {
		end.x = start.x + diff;
		end.y = start.y + diff;
		// if (x1,y1) is start and (x2,y2) is end then
		// new minC at top left i.e. x1, y1
		i = start.x; 
		j = start.y;
		minC = c[i*WINDOW_DIM + j];
		//new maxC at bottom right x2, y2
		i = end.x;
		j = end.y;
		maxC = c[i*WINDOW_DIM + j];
      	  }
      	  else if(y < start.y)		// bottom left to top right
          {
		end.x = start.x + diff;
		end.y = start.y - diff;
		//new minC at top left of box i.e. x1,y2 
		i = start.x;
		j = end.y;
		minC = c[i*WINDOW_DIM + j];
		//new maxC at bottom right of box i.e. x2,y1
		i = end.x;
		j = start.y;
		maxC = c[i*WINDOW_DIM + j];   	
	  }
    	}
    
	else if(x < start.x)
    	{
      	  if(y > start.y)		// top right to bottom left
      	  {
		end.x = start.x - diff;
		end.y = start.y + diff;

		i = end.x;
		j = start.y;
		minC = c[i*WINDOW_DIM + j];

		i = start.x;
		j = end.y;
		maxC = c[i*WINDOW_DIM + j];     	
      	  }
      	  else if (y < start.y)		// bottom right to top left
      	  {
		end.x = start.x - diff;
		end.y = start.y - diff;
 		
		i = end.x;
		j = end.y;
		minC = c[i*WINDOW_DIM + j];

		i = start.x;
		j = start.y;
		maxC = c[i*WINDOW_DIM + j];  	    
	  } 

    	} 
	
	drawMBSet();
	cout << "Zooming in. Level " << ++zoomLevel << endl;
	isSqrDrawn = false;
	glutPostRedisplay();    
   
  }
}

void motion(int x, int y)
{
  
  dx = abs(x - start.x);
  dy = abs(y - start.y);
 
  // Choosing smaller of the two diffs to get side of square
  if(dy > dx) 
    diff = dx;
  else 
    diff = dy;
  
  // Calculating new 'end' co-ordinates based on this diff
  if(x > start.x)
  {
    if(y > start.y)  		// top left to bottom right
    {
	end.x = start.x + diff;
	end.y = start.y + diff;
    }
    else if(y < start.y)	// bottom left to top right
    {
	end.x = start.x + diff;
	end.y = start.y - diff;
    }
  }
  else if(x < start.x)
  {
    if(y > start.y)		// top right to bottom left
    {
	end.x = start.x - diff;
	end.y = start.y + diff;
    }
    else if (y < start.y)	// bottom right to top left
    {
	end.x = start.x - diff;
	end.y = start.y - diff;
    }

  }

  glutPostRedisplay();
 
}

void keyboard(unsigned char key, int x, int y)
{
  if(key == 'B' || key == 'b')
  {
    if(!memStack.empty())
    {
	cout << "Zooming out. Level " << --zoomLevel << endl;

	minC = memStack.top().first;
	maxC = memStack.top().second;
	memStack.pop();	

	//recalculate MBSet with previous values of minC and maxC
	drawMBSet();
	glutPostRedisplay();
    }
    else
    {
	cout << "Already at maximum zoom out" << endl;
    }
  }

}

__host__ void clean()
{
  free(c);
  hipFree(dev_minC);
  hipFree(dev_maxC);
  hipFree(dev_c);
  hipFree(dev_iterations);
}

void init()
{
  glClearColor(0, 0, 0, 0);
  glViewport(0, 0, WINDOW_DIM, WINDOW_DIM);
  glMatrixMode(GL_PROJECTION);
  glLoadIdentity();
  glOrtho(0, WINDOW_DIM, WINDOW_DIM, 0, -WINDOW_DIM, WINDOW_DIM);
  glMatrixMode(GL_MODELVIEW);
  glLoadIdentity();
}

int main(int argc, char** argv)
{
  // Initialize OPENGL here
  // Set up necessary host and device buffers
  // set up the opengl callbacks for display, mouse and keyboard

  // Calculate the interation counts
  // Grad students, pick the colors for the 0 .. 1999 iteration count pixels
  atexit(clean);

  glutInit(&argc, argv);
  InitializeColors();	// calculate color array based on iteration counts
  drawMBSet();		//calculate MBSet

  glutInitDisplayMode(GLUT_DEPTH | GLUT_DOUBLE | GLUT_RGB);
  glutInitWindowSize(WINDOW_DIM, WINDOW_DIM);
  glutInitWindowPosition(200, 200);
  glutCreateWindow("Mandelbrot Set");

  init();  
  glutDisplayFunc(display);
  glutMouseFunc(mouse);
  glutMotionFunc(motion);
  glutKeyboardFunc(keyboard);
  
  glutMainLoop(); // THis will callback the display, keyboard and mouse

  return 0;
  
}
